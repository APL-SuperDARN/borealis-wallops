#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <zmq.hpp> // REVIEW #4 Need to explain what we use from this lib in our general documentation
#include <thread>
#include <complex>
#include <iostream>
#include <fstream>
#include <chrono>
#include <stdint.h>
#include <signal.h>
#include <cstdlib>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include "utils/protobuf/computationpacket.pb.h"
#include "utils/protobuf/sigprocpacket.pb.h"
#include "utils/driver_options/driveroptions.hpp"
#include "utils/signal_processing_options/signalprocessingoptions.hpp"
#include "utils/shared_memory/shared_memory.hpp"

#include "dsp.hpp"

extern "C" {
    #include "remez.h"
}

/*#define T_DEVICE_V(x) thrust::device_vector<x>
#define T_HOST_V(x) thrust::host_vector<x,thrust::cuda::experimental::pinned_allocator<x>>
#define T_COMPLEX_F thrust::complex<float>

#define sig_options.get_first_stage_sample_rate() 1.0e6 //1 MHz
#define sig_options.get_second_stage_sample_rate() 0.1e6 // 100 kHz
#define sig_options.get_third_stage_sample_rate() (10000.0/3.0) //3.33 kHz

#define sig_options.get_first_stage_filter_cutoff() 1.0e6
#define sig_options.get_first_stage_filter_transition() (sig_options.get_first_stage_filter_cutoff() * 0.5)

#define sig_options.get_second_stage_filter_cutoff() 0.1e6
#define sig_options.get_second_stage_filter_transition() (sig_options.get_second_stage_filter_cutoff() * 0.5)

#define sig_options.get_third_stage_filter_cutoff() (10000.0/3.0)
#define sig_options.get_third_stage_filter_transition() (sig_options.get_third_stage_filter_cutoff() * 0.25)
*/

// REVIEW #30 all filter-building functions could be placed in a separate file? 
std::vector<double> create_normalized_lowpass_filter_bands(float cutoff, float transition_band,
                        float Fs) { //REVIEW #5 units for params
    std::vector<double> filterbands; //REVIEW #3 describe band choices and how this works
    filterbands.push_back(0.0);
    filterbands.push_back(cutoff/Fs);
    filterbands.push_back((cutoff + transition_band)/Fs);
    filterbands.push_back(0.5);

    return filterbands;
}

// REVIEW #30 Is this in the right file? Should it be in dsp.cu along with get_gpu_properties?
// REVIEW #4 not sure where hipDeviceProp_t is found...
void print_gpu_properties(std::vector<hipDeviceProp_t> gpu_properties) {
    for(auto i : gpu_properties) { // REVIEW #28 does this need to be "auto&" stackoverflow says use this way "auto" if you aren't changing anything, but use auto& if you are.? 
        std::cout << "Device name: " << i.name << std::endl;
        std::cout << "  Max grid size x: " << i.maxGridSize[0] << std::endl;
        std::cout << "  Max grid size y: " << i.maxGridSize[1] << std::endl;
        std::cout << "  Max grid size z: " << i.maxGridSize[2] << std::endl;
        std::cout << "  Max threads per block: " << i.maxThreadsPerBlock
            << std::endl;
        std::cout << "  Max size of block dimension x: " << i.maxThreadsDim[0]
                << std::endl;
        std::cout << "  Max size of block dimension y: " << i.maxThreadsDim[1]
                << std::endl;
        std::cout << "  Max size of block dimension z: " << i.maxThreadsDim[2]
                << std::endl;
        std::cout << "  Memory Clock Rate (GHz): " << i.memoryClockRate/1e6
            << std::endl;
        std::cout << "  Memory Bus Width (bits): " << i.memoryBusWidth
            << std::endl;
        std::cout << "  Peak Memory Bandwidth (GB/s): " <<
           2.0*i.memoryClockRate*(i.memoryBusWidth/8)/1.0e6 << std::endl; // REVIEW #29 magic calculation with magic numbers? 
        std::cout << "  Max shared memory per block: " << i.sharedMemPerBlock
            << std::endl;
    }
}

uint32_t calculate_num_filter_taps(float rate, float transition_width) { // REVIEW #26 transition_width and transition_band are both used?
    auto k = 3; //from formula 7-6 of Lyons text REVIEW #1 and #7 add some explanation for design of choice of k=3
    auto num_taps = k * (rate/transition_width); //REVIEW #5 provide units for rate & transition width

    //The parallel reduction in the GPU code requires at least 64 taps,
    //so we have to use at least that many as a minimum.
    return (num_taps > 64) ? num_taps : 64;
    //return num_taps; // REVIEW #33
    //REVIEW #28 this function is returning a uint32; will this always round down? comment/document this
}

std::vector<std::complex<float>> create_filter(uint32_t num_taps, float filter_cutoff, float transition_width, // REVIEW #31 this line is over 80 characters, need to be consistent. #31
                                    float rate) { // REVIEW #5 filter_cutoff, transition_width, rate
    // REVIEW #3 explain algorithm including weight
    std::vector<double> desired_band_gain = {1.0,0.0}; 
    std::vector<double> weight = {1.0,1.0}; 

    auto filter_bands = create_normalized_lowpass_filter_bands(filter_cutoff,transition_width,
                            rate);

    /*remez returns number of taps + 1. Should we use num_taps + 1
      or should we pass num_taps - 1 to remez?
    */ //REVIEW #6 make this a TODO - does this depend on num_taps being odd or even?
    double filter_taps[num_taps + 1]; 
    auto converges = remez(filter_taps, num_taps + 1, (filter_bands.size()/2), // REVIEW #32 declare and initialize number of bands for better readability
        filter_bands.data(),desired_band_gain.data(),weight.data(),BANDPASS,GRIDDENSITY); // REVIEW #15 are we passing the right values, should error check before passing these params
    if (converges < 0){ // REVIEW #10 remez returns True or False is this check correct?
        std::cerr << "Filter failed to converge with cutoff of " << filter_cutoff // REVIEW #5 & #34 print units
            << ", transition width " << transition_width << ", and rate "
            << rate << std::endl;
        //TODO(keith): throw error
    }

    std::vector<std::complex<float>> complex_taps(num_taps + 1);
    for (auto &i : filter_taps) {
        complex_taps[i] = std::complex<float>(i,0.0); // REVIEW #0 does this work?
    }

    return complex_taps;
}

void save_filter_to_file(std::vector<std::complex<float>> filter_taps, const char* name) {
    std::ofstream filter;
    filter.open(name);
    for (auto &i : filter_taps){
        filter << i << std::endl;
    }
    filter.close();
}


int main(int argc, char **argv){
    // REVIEW #35 main is > 200 lines, kind of large maybe the entire filter setup process could be moved out?
    GOOGLE_PROTOBUF_VERIFY_VERSION; // REVIEW #4 state what this does? macro to verify headers and lib are same version.

    auto driver_options = DriverOptions();
    auto sig_options = SignalProcessingOptions(); // #26 REVIEW Should the naming be updated along with DSP ?
    auto rx_rate = driver_options.get_rx_rate(); // #5 REVIEW What units is rx_rate in? 
    zmq::context_t sig_proc_context(1); // REVIEW #4 - what is "1"?

    zmq::socket_t driver_socket(sig_proc_context, ZMQ_PAIR);
    driver_socket.bind("ipc:///tmp/feeds/1"); // REVIEW #29 Should this be in a config file? Sort of a magic string right now

    // REVIEW #1 Need a comment here to explain which blocks these 3 sockets talk to, the driver socket is obvious, but these three may not be
    zmq::socket_t radarctrl_socket(sig_proc_context, ZMQ_PAIR); // REVIEW #26 Name of radarctrl may need to be updated to be consistent with our discussion on Friday March 10th
    radarctrl_socket.bind("ipc:///tmp/feeds/2");

    zmq::socket_t ack_socket(sig_proc_context, ZMQ_PAIR);
    ack_socket.bind("ipc:///tmp/feeds/3");

    zmq::socket_t timing_socket(sig_proc_context, ZMQ_PAIR);
    timing_socket.bind("ipc:///tmp/feeds/4");

    auto gpu_properties = get_gpu_properties(); 
    print_gpu_properties(gpu_properties);

    uint32_t first_stage_dm_rate, second_stage_dm_rate, third_stage_dm_rate = 0; // REVIEW #32 this line only initializes third stage, the others are only declared. bad style?
    if (fmod(rx_rate,sig_options.get_first_stage_sample_rate()) > 0.0){ // REVIEW #1 or #2 perhaps a comment needed to clarify what this error means (dm rate needs to be int)
        //TODO(keith): handle error
    }
    else{
        auto rate_f = rx_rate/sig_options.get_first_stage_sample_rate(); // REVIEW #26 don't like the name rate_f, not obvious why it's called that, seems like a temp variable, so perhaps indicate that?
        first_stage_dm_rate = static_cast<uint32_t>(rate_f);

        rate_f = sig_options.get_first_stage_sample_rate()/
                    sig_options.get_second_stage_sample_rate();
        second_stage_dm_rate = static_cast<uint32_t>(rate_f);

        rate_f = sig_options.get_second_stage_sample_rate()/
                    sig_options.get_third_stage_sample_rate();
        third_stage_dm_rate = static_cast<uint32_t>(rate_f);
    }
    // REVIEW #15 Even though the second and third stage sample rates are set in config, the same error check should be done on the 2nd and 3rd stage dm rates as for first stage dm rate.

    std::cout << "1st stage dm rate: " << first_stage_dm_rate << std::endl
        << "2nd stage dm rate: " << second_stage_dm_rate << std::endl
        << "3rd stage dm rate: " << third_stage_dm_rate << std::endl;


    auto S_lowpass1 = calculate_num_filter_taps(rx_rate, //REVIEW #26 perhaps name of S_lowpass1 could be more self-explanatory as lowpass1_numtaps ?
                                    sig_options.get_first_stage_filter_transition());
    auto S_lowpass2 = calculate_num_filter_taps(sig_options.get_first_stage_sample_rate(),
                                    sig_options.get_second_stage_filter_transition());
    auto S_lowpass3 = calculate_num_filter_taps(sig_options.get_second_stage_sample_rate(),
                                    sig_options.get_third_stage_filter_transition());

    std::cout << "1st stage taps: " << S_lowpass1 << std::endl << "2nd stage taps: " // REVIEW #34 mention that it's the number of taps?
        << S_lowpass2 << std::endl << "3rd stage taps: " << S_lowpass3 <<std::endl;


    std::chrono::steady_clock::time_point timing_start = std::chrono::steady_clock::now();


    auto filtertaps_1 = create_filter(S_lowpass1, sig_options.get_first_stage_filter_cutoff(),
                        sig_options.get_first_stage_filter_transition(), rx_rate);
    auto filtertaps_2 = create_filter(S_lowpass2,sig_options.get_second_stage_filter_cutoff(),
                        sig_options.get_second_stage_filter_transition(),
                        sig_options.get_first_stage_sample_rate());
    auto filtertaps_3 = create_filter(S_lowpass3,sig_options.get_third_stage_filter_cutoff(),
                        sig_options.get_third_stage_filter_transition(),
                        sig_options.get_second_stage_sample_rate());

    std::chrono::steady_clock::time_point timing_end = std::chrono::steady_clock::now();
    std::cout << "Time to create 3 filters: "
      << std::chrono::duration_cast<std::chrono::microseconds>
                                                  (timing_end - timing_start).count()
      << "us" << std::endl;

    save_filter_to_file(filtertaps_1,"filter1coefficients.dat");
    save_filter_to_file(filtertaps_2,"filter2coefficients.dat");
    save_filter_to_file(filtertaps_3,"filter3coefficients.dat");

    while(1){
        //Receive packet from radar control
        zmq::message_t radctl_request;
        radarctrl_socket.recv(&radctl_request);
        sigprocpacket::SigProcPacket sp;
        std::string r_msg_str(static_cast<char*>(radctl_request.data()), radctl_request.size());
        sp.ParseFromString(r_msg_str);

        std::cout << "Got radarctrl request" << std::endl;

        //Then receive packet from driver
        zmq::message_t driver_request;
        driver_socket.recv(&driver_request);
        computationpacket::ComputationPacket cp;
        std::string c_msg_str(static_cast<char*>(driver_request.data()), driver_request.size());
        cp.ParseFromString(c_msg_str);

        std::cout << "Got driver request" << std::endl;

        //Verify driver and radar control packets align
        if (sp.sequence_num() != cp.sequence_num()) {
            //TODO(keith): handle error
            std::cout << "SEQUENCE NUMBER mismatch rctl: " << sp.sequence_num()
                << " driver: " << cp.sequence_num();
        }


        //Receive driver samples now
        //timing_start = std::chrono::steady_clock::now();
        //driver_socket.recv(&driver_request);
        //timing_end = std::chrono::steady_clock::now();
        //std::cout << "recv: "
        //  << std::chrono::duration_cast<std::chrono::microseconds>(timing_end - timing_start).count()
        //  << "us" << std::endl;

        //auto start = static_cast<T_COMPLEX_F *>(driver_request.data());
        //auto data_size = static_cast<size_t>(driver_request.size());
        //auto num_elements = data_size/sizeof(T_COMPLEX_F);



        //Parse needed packet values now
        std::vector<double> rx_freqs;
        for(int i=0; i<sp.rxchannel_size(); i++) {
            rx_freqs.push_back(sp.rxchannel(i).rxfreq());
        }

        timing_start = std::chrono::steady_clock::now();

        std::vector<std::complex<float>> filtertaps_1_bp_h(rx_freqs.size()*filtertaps_1.size());
        for (int i=0; i<rx_freqs.size(); i++) {
            auto sampling_freq = 2 * M_PI * rx_freqs[i]/rx_rate;

            for(int j=0;j < filtertaps_1.size(); j++) {
                auto radians = fmod(sampling_freq * j,2 * M_PI);
                auto I = filtertaps_1[j].real() * cos(radians);
                auto Q = filtertaps_1[j].real() * sin(radians);
                filtertaps_1_bp_h[i*filtertaps_1.size() + j] = std::complex<float>(I,Q);
            }
        }

        timing_end = std::chrono::steady_clock::now();

        std::cout << "NCO mix timing: "
          << std::chrono::duration_cast<std::chrono::microseconds>(timing_end - timing_start).count()
          << "us" << std::endl;

        std::vector<std::complex<float>> filtertaps_2_h(filtertaps_2.size());
        std::vector<std::complex<float>> filtertaps_3_h(filtertaps_3.size());
        for (uint32_t i=0; i< rx_freqs.size(); i++){
            filtertaps_2_h.insert(filtertaps_2_h.end(),filtertaps_2.begin(),filtertaps_2.end());
            filtertaps_3_h.insert(filtertaps_3_h.end(),filtertaps_3.begin(),filtertaps_3.end());
        }

        DSPCore *dp = new DSPCore(&ack_socket, &timing_socket,
                                                         sp.sequence_num(), cp.name().c_str());


        auto total_antennas = sig_options.get_main_antenna_count() +
                                sig_options.get_interferometer_antenna_count();
        auto total_samples = cp.numberofreceivesamples() * total_antennas;

        std::cout << "Total elements in data message: " << total_samples
            << std::endl;

        dp->allocate_and_copy_rf_samples(total_samples);
        dp->allocate_and_copy_first_stage_filters(filtertaps_1_bp_h.data(), filtertaps_1_bp_h.size());


        auto num_output_samples_1 = rx_freqs.size() * cp.numberofreceivesamples()/first_stage_dm_rate
                                        * total_antennas;
        dp->allocate_first_stage_output(num_output_samples_1);

        gpuErrchk(hipStreamAddCallback(dp->get_cuda_stream(),
                                    DSPCore::initial_memcpy_callback, dp, 0));

        dp->call_decimate(dp->get_rf_samples_p(),
            dp->get_first_stage_output_p(),
            dp->get_first_stage_bp_filters_p(), first_stage_dm_rate,
            cp.numberofreceivesamples(), filtertaps_1.size(), rx_freqs.size(),
            total_antennas, "First stage of decimation");



        dp->allocate_and_copy_second_stage_filters(filtertaps_2_h.data(), filtertaps_2_h.size());
        auto num_output_samples_2 = num_output_samples_1 / second_stage_dm_rate;
        dp->allocate_second_stage_output(num_output_samples_2);
        auto num_samps_2 = cp.numberofreceivesamples()/first_stage_dm_rate;
        dp->call_decimate(dp->get_first_stage_output_p(),
            dp->get_second_stage_output_p(),
            dp->get_second_stage_filters_p(), second_stage_dm_rate,
            num_samps_2, filtertaps_2.size(), rx_freqs.size(),
            total_antennas, "Second stage of decimation");



        dp->allocate_and_copy_third_stage_filters(filtertaps_3_h.data(), filtertaps_3_h.size());
        auto num_output_samples_3 = num_output_samples_2 / third_stage_dm_rate;
        dp->allocate_third_stage_output(num_output_samples_3);
        auto num_samps_3 = num_samps_2/second_stage_dm_rate;
        dp->call_decimate(dp->get_second_stage_output_p(),
            dp->get_third_stage_output_p(),
            dp->get_third_stage_filters_p(), third_stage_dm_rate,
            num_samps_3, filtertaps_3.size(), rx_freqs.size(),
            total_antennas, "Third stage of decimation");

        dp->allocate_and_copy_host_output(num_output_samples_3);

        // New in CUDA 5.0: Add a CPU callback which is called once all currently pending operations in the CUDA stream have finished
        gpuErrchk(hipStreamAddCallback(dp->get_cuda_stream(),
                                            DSPCore::cuda_postprocessing_callback, dp, 0));


    }


}
