#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <zmq.hpp> // REVIEW #4 Need to explain what we use from this lib in our general documentation
#include <thread>
#include <complex>
#include <iostream>
#include <fstream>
#include <chrono>
#include <stdint.h>
#include <signal.h>
#include <cstdlib>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include "utils/protobuf/rxsamplesmetadata.pb.h"
#include "utils/protobuf/sigprocpacket.pb.h"
#include "utils/driver_options/driveroptions.hpp"
#include "utils/signal_processing_options/signalprocessingoptions.hpp"
#include "utils/shared_memory/shared_memory.hpp"

#include "dsp.hpp"
#include "filtering.hpp"
#include "decimate.hpp"

#ifdef DEBUG
#define DEBUG_MSG(x) do {std::cerr << x << std::endl;} while (0)
#else
#define DEBUG_MSG(x)
#endif

#define ERR_CHK_ZMQ(x) try {x;} catch (zmq::error_t& e) {} //TODO(keith): handle error


int main(int argc, char **argv){
  GOOGLE_PROTOBUF_VERIFY_VERSION; // Verifies that header and lib are same version.

  //TODO(keith): verify config options.
  auto driver_options = DriverOptions();
  auto sig_options = SignalProcessingOptions();
  auto rx_rate = driver_options.get_rx_rate(); //Hz

  zmq::context_t sig_proc_context(1); // 1 is context num. Only need one per program as per examples

  zmq::socket_t driver_socket(sig_proc_context, ZMQ_PAIR);
  ERR_CHK_ZMQ(driver_socket.bind("ipc:///tmp/feeds/1"))


  //This socket is used to receive metadata about the sequence to process
  zmq::socket_t radar_control_socket(sig_proc_context, ZMQ_PAIR);
  ERR_CHK_ZMQ(radar_control_socket.bind("ipc:///tmp/feeds/2"))

  //This socket is used to acknowledge a completed sequence to radar_control
  zmq::socket_t ack_socket(sig_proc_context, ZMQ_PAIR);
  ERR_CHK_ZMQ(ack_socket.bind("ipc:///tmp/feeds/3"))

  //This socket is used to send the GPU kernel timing to radar_control to know if the processing
  //can be done in real-time.
  zmq::socket_t timing_socket(sig_proc_context, ZMQ_PAIR);
  ERR_CHK_ZMQ(timing_socket.bind("ipc:///tmp/feeds/4"))


  auto gpu_properties = get_gpu_properties();
  print_gpu_properties(gpu_properties);

  uint32_t first_stage_dm_rate = 0, second_stage_dm_rate = 0, third_stage_dm_rate = 0;
  //Check for non integer dm rates
  if (fmod(rx_rate,sig_options.get_first_stage_sample_rate()) > 0.0) {
    //TODO(keith): handle error
  } //TODO(keith): not sure these checks will work.
/*  else if (fmod(sig_options.get_first_stage_sample_rate(),
          sig_options.get_second_stage_sample_rate()) > 0.0) {
    //TODO(keith): handle error
  }
  else if(fmod(sig_options.get_second_stage_sample_rate(),
        sig_options.get_third_stage_sample_rate()) > 0.0) {
    //TODO(keith): handle error
  }*/
  else{
    auto float_dm_rate = rx_rate/sig_options.get_first_stage_sample_rate();
    first_stage_dm_rate = static_cast<uint32_t>(float_dm_rate);

    float_dm_rate = sig_options.get_first_stage_sample_rate()/
          sig_options.get_second_stage_sample_rate();
    second_stage_dm_rate = static_cast<uint32_t>(float_dm_rate);

    float_dm_rate = sig_options.get_second_stage_sample_rate()/
          sig_options.get_third_stage_sample_rate();
    third_stage_dm_rate = static_cast<uint32_t>(float_dm_rate);
  }

  std::cout << "1st stage dm rate: " << first_stage_dm_rate << std::endl
    << "2nd stage dm rate: " << second_stage_dm_rate << std::endl
    << "3rd stage dm rate: " << third_stage_dm_rate << std::endl;


  auto filter_timing_start = std::chrono::steady_clock::now();

  Filtering filters(rx_rate,sig_options);

  DEBUG_MSG("Number of 1st stage taps: " << filters.get_num_first_stage_taps() << std::endl
    << "Number of 2nd stage taps: " << filters.get_num_second_stage_taps() << std::endl
    << "Number of 3rd stage taps: " << filters.get_num_third_stage_taps() <<std::endl
    << "Number of 1st stage taps after padding: "
    << filters.get_first_stage_lowpass_taps().size() << std::endl
    << "Number of 2nd stage taps after padding: "
    << filters.get_second_stage_lowpass_taps().size() << std::endl
    << "Number of 3rd stage taps after padding: "
    << filters.get_third_stage_lowpass_taps().size());

  auto filter_timing_end = std::chrono::steady_clock::now();
  auto time_diff = std::chrono::duration_cast<std::chrono::microseconds>(filter_timing_end -
                                                                       filter_timing_start).count();
  DEBUG_MSG("Time to create 3 filters: " << time_diff << "us");

  //FIXME(Keith): fix saving filter to file
  filters.save_filter_to_file(filters.get_first_stage_lowpass_taps(),"filter1coefficients.dat");
  filters.save_filter_to_file(filters.get_second_stage_lowpass_taps(),"filter2coefficients.dat");
  filters.save_filter_to_file(filters.get_third_stage_lowpass_taps(),"filter3coefficients.dat");

  while(1){
    //Receive packet from radar control
    zmq::message_t radctl_request;
    radar_control_socket.recv(&radctl_request);
    sigprocpacket::SigProcPacket sp_packet;
    std::string radctrl_str(static_cast<char*>(radctl_request.data()), radctl_request.size());
    if (sp_packet.ParseFromString(radctrl_str) == false){
      //TODO(keith): handle error
    }

    //Then receive packet from driver
    zmq::message_t driver_request;
    driver_socket.recv(&driver_request);
    rxsamplesmetadata::RxSamplesMetadata rx_metadata;
    std::string driver_str(static_cast<char*>(driver_request.data()), driver_request.size());
    if (rx_metadata.ParseFromString(driver_str) == false) {
      //TODO(keith): handle error
    }

    DEBUG_MSG("Got driver request");

    //Verify driver and radar control packets align
    if (sp_packet.sequence_num() != rx_metadata.sequence_num()) {
      //TODO(keith): handle error
      DEBUG_MSG("SEQUENCE NUMBER mismatch radar_control: " << sp_packet.sequence_num()
        << " usrp_driver: " << rx_metadata.sequence_num());
    }

    //Parse needed packet values now
    if (sp_packet.rxchannel_size() == 0) {
      //TODO(keith): handle error
    }
    std::vector<double> rx_freqs;
    for(int i=0; i<sp_packet.rxchannel_size(); i++) {
      rx_freqs.push_back(sp_packet.rxchannel(i).rxfreq());
    }

    auto mix_timing_start = std::chrono::steady_clock::now();

    filters.mix_first_stage_to_bandpass(rx_freqs,rx_rate);

    auto mix_timing_end = std::chrono::steady_clock::now();

    time_diff = std::chrono::duration_cast<std::chrono::microseconds>(mix_timing_end -
                                                                        mix_timing_start).count();

    DEBUG_MSG("NCO mix timing: " << time_diff<< "us");

    if (rx_metadata.shrmemname().empty()){
      //TODO(keith): handle missing name error
    }
    DSPCore *dp = new DSPCore(&ack_socket, &timing_socket,
                             sp_packet.sequence_num(), rx_metadata.shrmemname());


    auto total_antennas = sig_options.get_main_antenna_count() +
                sig_options.get_interferometer_antenna_count();

    if (rx_metadata.numberofreceivesamples() == 0){
      //TODO(keith): handle error for missing number of samples.
    }
    auto total_samples = rx_metadata.numberofreceivesamples() * total_antennas;

    DEBUG_MSG("Total samples in data message: " << total_samples);

    dp->allocate_and_copy_rf_samples(total_samples);
    dp->allocate_and_copy_first_stage_filters(filters.get_first_stage_bandpass_taps_h().data(),
                                                filters.get_first_stage_bandpass_taps_h().size());

    auto num_output_samples_1 = rx_freqs.size() *
                                  (rx_metadata.numberofreceivesamples()/first_stage_dm_rate) *
                                  total_antennas;

    dp->allocate_first_stage_output(num_output_samples_1);

    gpuErrchk(hipStreamAddCallback(dp->get_cuda_stream(),
                  DSPCore::initial_memcpy_callback, dp, 0));

    call_decimate<DecimationType::bandpass>(dp->get_rf_samples_p(),
      dp->get_first_stage_output_p(),
      dp->get_first_stage_bp_filters_p(), first_stage_dm_rate,
      rx_metadata.numberofreceivesamples(), filters.get_first_stage_lowpass_taps().size(), rx_freqs.size(),
      total_antennas, "First stage of decimation", dp->get_cuda_stream());


    // When decimating, we go from one set of samples for each antenna in the first stage
    // to multiple sets of reduced samples for each frequency in further stages. Output samples are
    // grouped by frequency with all samples for each antenna following each other
    // before samples of another frequency start. In the first stage need a filter for each 
    // frequency, but in the next stages we only need one filter for all data sets.
    dp->allocate_and_copy_second_stage_filter(filters.get_second_stage_lowpass_taps().data(),
                                                filters.get_second_stage_lowpass_taps().size());

    auto num_output_samples_2 = num_output_samples_1 / second_stage_dm_rate;

    dp->allocate_second_stage_output(num_output_samples_2);

    // each antenna has a data set for each frequency after filtering.
    auto samples_per_antenna_2 = num_output_samples_1/total_antennas;
    call_decimate<DecimationType::lowpass>(dp->get_first_stage_output_p(),
      dp->get_second_stage_output_p(),
      dp->get_second_stage_filter_p(), second_stage_dm_rate,
      samples_per_antenna_2, filters.get_second_stage_lowpass_taps().size(), rx_freqs.size(),
      total_antennas, "Second stage of decimation", dp->get_cuda_stream());


    dp->allocate_and_copy_third_stage_filter(filters.get_third_stage_lowpass_taps().data(),
                                               filters.get_third_stage_lowpass_taps().size());
    auto num_output_samples_3 = num_output_samples_2 / third_stage_dm_rate;
    dp->allocate_third_stage_output(num_output_samples_3);
    auto samples_per_antenna_3 = samples_per_antenna_2/second_stage_dm_rate;
    call_decimate<DecimationType::lowpass>(dp->get_second_stage_output_p(),
      dp->get_third_stage_output_p(),
      dp->get_third_stage_filter_p(), third_stage_dm_rate,
      samples_per_antenna_3, filters.get_third_stage_lowpass_taps().size(), rx_freqs.size(),
      total_antennas, "Third stage of decimation", dp->get_cuda_stream());

    dp->allocate_and_copy_host_output(num_output_samples_3);

    gpuErrchk(hipStreamAddCallback(dp->get_cuda_stream(),
                      DSPCore::cuda_postprocessing_callback, dp, 0));

    hipDeviceSynchronize();

  }



}
