
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <stdint.h>

/*Overloaded __shfl_down function. Default does not recognize cuComplex but
does for equivalent float2 type.
https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
http://docs.nvidia.com/cuda/cuda-c-programming-guide/#warp-shuffle-functions
*/
__device__ inline hipComplex __shfl_down(hipComplex var, unsigned int srcLane, int width=32){
    float2 a = *reinterpret_cast<float2*>(&var);
    a.x = __shfl_down(a.x, srcLane, width); // REVIEW #0 Does this call the original function since the a variable is a float now?
    a.y = __shfl_down(a.y, srcLane, width);
    return *reinterpret_cast<hipComplex*>(&a);
}

/*Slightly modified version of reduction #5 from NVIDIA examples
/usr/local/cuda/samples/6_Advanced/reduction
*/
__device__ hipComplex parallel_reduce(hipComplex* data, int tap_offset) { // REVIEW #28 can tap_offset ever be negative? Maybe should make it uint32_t

    auto filter_tap_num = threadIdx.x;
    auto num_filter_taps = blockDim.x;
    hipComplex total_sum = data[tap_offset];


    if ((num_filter_taps >= 512) && (filter_tap_num < 256))
    {
        data[tap_offset] = total_sum = hipCaddf(total_sum,data[tap_offset  + 256]); // REVIEW #25 Is it necessary for speed to have two '=' statements on one line? it took a while to see the second one, therefore more confusing. split into two lines
    }

    __syncthreads();

    if ((num_filter_taps >= 256) && (filter_tap_num < 128))
    {
            data[tap_offset] = total_sum = hipCaddf(total_sum, data[tap_offset + 128]);
    }

     __syncthreads();

    if ((num_filter_taps >= 128) && (filter_tap_num <  64))
    {
       data[tap_offset] = total_sum = hipCaddf(total_sum, data[tap_offset  +  64]);
    }

    __syncthreads();

    if ( filter_tap_num < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (num_filter_taps >=  64) total_sum = hipCaddf(total_sum, data[tap_offset + 32]);
        // Reduce final warp using shuffle // REVEW #3 This code depends upon a warp all executing threads at exactly the same time, if it didn't then double the total_sum value for the second half of the threads would be accidentally used. Can be explicit by putting if statement in the for loop [if (filter_tap_num < offset) we think]
        for (int offset = warpSize/2; offset > 0; offset /= 2) // REVIEW #0 Where does warpSize come from? Don't you need to get it from the gpu_properties?
        {
            total_sum = hipCaddf(total_sum,__shfl_down(total_sum, offset)); // REVIEW #3 Very not-obvious. Seems like it needs to know that total_sum is the variable/memory to work on, need a comment to tell us how this works
        }
    }

    return total_sum;
}

__global__ void decimate1024(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel) //REVIEW #1 describe thread/block/grid dimensions and indices 
{ 

    extern __shared__ hipComplex filter_products[]; // REVIEW #4 comment why is this extern and why is it necessary to be dynamically allocated?

    auto channel_num = blockIdx.y;
    auto channel_offset = channel_num * samples_per_channel;

    auto dec_sample_num = blockIdx.x;
    auto dec_sample_offset = dec_sample_num * dm_rate;

    auto tap_offset = threadIdx.y * blockDim.y + threadIdx.x; // REVIEW #0 should be blockDim.x

    hipComplex sample;
    if ((dec_sample_offset + threadIdx.x) >= samples_per_channel) {
        sample = make_hipComplex(0.0,0.0); // REVIEW #1 explain zero-padding, #0, correct this after to throw out edge effects (per stage) ceil((num_samps - num_taps)/dm_rate)
    }
    else {
        auto final_offset = channel_offset + dec_sample_offset + threadIdx.x;
        sample = original_samples[final_offset];
    }


    filter_products[tap_offset] = hipCmulf(sample,filter_taps[tap_offset]); // REVIEW #4 tell user that this comes from cuComplex.h, any side effects?

    __syncthreads(); // REVIEW #1 Synchronizes all threads in a block, meaning 1 output sample per rx freq is ready to be calculated with the parallel reduce

// REVIEW #33 Use git to keep old code if you need it
/*    auto num_taps = blockDim.x;
    for (unsigned int s=num_taps/2; s>32; s>>=1) {
        if (threadIdx.x < s)
            filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                    filter_products[tap_offset + s]);
        __syncthreads();
    }
    if (threadIdx.x < 32){
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 32]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 16]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 8]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 4]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 2]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 1]);
        __syncthreads();
    }
*/
    auto total_sum = parallel_reduce(filter_products, tap_offset); // REVIEW #26 Should this be called something like 'decimated_sample' instead - to indicate that it is going into the array of decimated samples? total_sum could be the variable name in parallel reduce, but not here in this context

    if (threadIdx.x == 0) { // REVIEW #1 Explain how you're setting up the array of decimated samples
        channel_offset = channel_num * samples_per_channel/dm_rate; // REVIEW #13 gridDimx is already samples_per_channel/dm_rate, use it instead
        auto total_channels = blockDim.y; // REVIEW #0 This should be gridDim.y if you intend to use 'total_channels' as antennas (should use 'antennas')
        auto freq_offset = threadIdx.y * total_channels; // REVIEW #0 still need to multiply by gridDim.x here to get index into proper location
        auto total_offset = freq_offset + channel_offset + dec_sample_num;
        decimated_samples[total_offset] = total_sum;//filter_products[tap_offset];
    }
}

__global__ void decimate2048(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel)
{

    extern __shared__ hipComplex filter_products[];

    auto channel_num = blockIdx.y; // REVIEW #26 -Again here channels/freqs/antennas is confused and needs to be consistent, maybe we avoid the word 'channel' altogether
    auto channel_offset = channel_num * samples_per_channel;

    auto dec_sample_num = blockIdx.x;
    auto dec_sample_offset = dec_sample_num * dm_rate;

    auto tap_offset = threadIdx.y * blockDim.y + 2 * threadIdx.x; //REVIEW #0 should be blockDim.x 

    hipComplex sample_1;
    hipComplex sample_2;
    if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_channel) { 
        sample_1 = make_hipComplex(0.0,0.0);
        sample_2 = make_hipComplex(0.0,0.0);
    }
    else {
        auto final_offset = channel_offset + dec_sample_offset + 2*threadIdx.x;
        sample_1 = original_samples[final_offset];
        sample_2 = original_samples[final_offset+1];  // REVIEW #0 what if final_offset = samples_per_channel - 1 so that sample_1 is in bounds but sample_2 is out of bounds
    }


    filter_products[tap_offset] = hipCmulf(sample_1,filter_taps[tap_offset]); //
    filter_products[tap_offset+1] = hipCmulf(sample_2, filter_taps[tap_offset+1]); // REVIEW #0 what if you have an odd number of taps so that in the last thread filter_taps[tap_offset+1] isn't defined ? (unless all filters are of length 2^x)
    // REVIEW #0 does parallel reduce work when you are only passing even tap_offset values? should we change filter_products to be half the length of filter_taps and do a filter_products[tap_offset/2] = cuCaddf(filter_products[tap_offset/2], cuCmulf(sample_2, filter_taps[tap_offset+1])) here?  suggest creating a new variable for offset within filter_products
    __syncthreads();

      // REVIEW # 33 use git
/*    auto half_num_taps = blockDim.x;
    for (unsigned int s=half_num_taps; s>32; s>>=1) {
        if (threadIdx.x < s)
            filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                    filter_products[tap_offset + s]);
        __syncthreads();
    }
    if (threadIdx.x < 32){
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 32]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 16]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 8]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 4]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 2]);
        __syncthreads();
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 1]);
        __syncthreads();
    }*/

/*    auto half_num_taps = blockDim.x;
    for (unsigned int s=half_num_taps; s>32; s>>=1) {
        if (tap_offset < s)
            filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                    filter_products[tap_offset + s]);
            filter_products[tap_offset+1] = cuCaddf(filter_products[tap_offset+1],
                                                        filter_products[tap_offset+1 + s]);
        __syncthreads();
    }
    if (tap_offset < 32){
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 32]);
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 16]);
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 8]);
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 4]);
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 2]);
        filter_products[tap_offset] = cuCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 1]);

        filter_products[tap_offset + 1] = cuCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 32]);
        filter_products[tap_offset + 1] = cuCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 16]);
        filter_products[tap_offset + 1] = cuCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 8]);
        filter_products[tap_offset + 1] = cuCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 4]);
        filter_products[tap_offset + 1] = cuCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 2]);
        filter_products[tap_offset + 1] = cuCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 1]);
    }*/

    auto total_sum = parallel_reduce(filter_products, tap_offset); // REVIEW #0 pass new variable for offset in filter products so you are not passing only even values
    if (threadIdx.x == 0) { // REVIEW #1 Explain how you're setting up the array of decimated samples
        channel_offset = channel_num * samples_per_channel/dm_rate; // REVIEW #13 gridDimx is already samples_per_channel/dm_rate, use it instead
        auto total_channels = blockDim.y; // REVIEW #0 This should be gridDim.y if you intend to use 'total_channels' as antennas (should use 'antennas')
        auto freq_offset = threadIdx.y * total_channels; // REVIEW #0 still need to multiply by gridDim.x here to get index into proper location
        auto total_offset = freq_offset + channel_offset + dec_sample_num;
        decimated_samples[total_offset] = total_sum;//filter_products[tap_offset];
    }
}

static dim3 create_grid(uint32_t num_samples, uint32_t dm_rate, uint32_t num_channels) // REVIEW #26 no more channels
{
    auto num_blocks_x = num_samples/dm_rate;
    auto num_blocks_y = num_channels;
    auto num_blocks_z = 1;
    std::cout << "    Grid size: " << num_blocks_x << " x " << num_blocks_y << " x "
        << num_blocks_z << std::endl;
    dim3 dimGrid(num_blocks_x,num_blocks_y,num_blocks_z);

    return dimGrid;
}

static dim3 create_block(uint32_t num_taps, uint32_t num_freqs)
{
    auto num_threads_x = num_taps;
    auto num_threads_y = num_freqs;
    auto num_threads_z = 1;
    std::cout << "    Block size: " << num_threads_x << " x " << num_threads_y << " x "
        << num_threads_z << std::endl;
    dim3 dimBlock(num_threads_x,num_threads_y,num_threads_z);

    return dimBlock;
}

void decimate1024_wrapper(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel, uint32_t num_taps, uint32_t num_freqs,
    uint32_t num_channels, hipStream_t stream) { // REVIEW #1 describe how this works including choice of blocks and grids

    auto shr_mem_taps = num_freqs * num_taps * sizeof(hipComplex); // REVIEW #32 why do we need this?
    std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

    auto dimGrid = create_grid(samples_per_channel, dm_rate, num_channels); 
    auto dimBlock = create_block(num_taps,num_freqs);
    decimate1024<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
                filter_taps, dm_rate, samples_per_channel);

}

void decimate2048_wrapper(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel, uint32_t num_taps, uint32_t num_freqs,
    uint32_t num_channels, hipStream_t stream) {

    auto shr_mem_taps = num_freqs * num_taps * sizeof(hipComplex);
    std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

    auto dimGrid = create_grid(samples_per_channel, dm_rate, num_channels);
    auto dimBlock = create_block(num_taps/2, num_freqs);
    decimate2048<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
        filter_taps, dm_rate, samples_per_channel);
}
