
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <stdint.h>

/*Overloaded __shfl_down function. Default does not recognize cuComplex but
does for equivalent float2 type.
https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
http://docs.nvidia.com/cuda/cuda-c-programming-guide/#warp-shuffle-functions
*/
__device__ inline hipComplex __shfl_down(hipComplex var, unsigned int srcLane, int width=32){
    float2 a = *reinterpret_cast<float2*>(&var);
    a.x = __shfl_down(a.x, srcLane, width);
    a.y = __shfl_down(a.y, srcLane, width);
    return *reinterpret_cast<hipComplex*>(&a);
}

/*Slightly modified version of reduction #5 from NVIDIA examples
/usr/local/cuda/samples/6_Advanced/reduction
*/
__device__ hipComplex parallel_reduce(hipComplex* data, int tap_offset) {

    auto filter_tap_num = threadIdx.x;
    auto num_filter_taps = blockDim.x;
    hipComplex total_sum = data[tap_offset];


    if ((num_filter_taps >= 512) && (filter_tap_num < 256))
    {
        data[tap_offset] = total_sum = hipCaddf(total_sum,data[tap_offset  + 256]);
    }

    __syncthreads();

    if ((num_filter_taps >= 256) && (filter_tap_num < 128))
    {
            data[tap_offset] = total_sum = hipCaddf(total_sum, data[tap_offset + 128]);
    }

     __syncthreads();

    if ((num_filter_taps >= 128) && (filter_tap_num <  64))
    {
       data[tap_offset] = total_sum = hipCaddf(total_sum, data[tap_offset  +  64]);
    }

    __syncthreads();

    if ( filter_tap_num < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (num_filter_taps >=  64) total_sum = hipCaddf(total_sum, data[tap_offset + 32]);
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            total_sum = hipCaddf(total_sum,__shfl_down(total_sum, offset));
        }
    }

    return total_sum;
}

__global__ void decimate1024(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel)
{

    extern __shared__ hipComplex filter_products[];

    auto channel_num = blockIdx.y;
    auto channel_offset = channel_num * samples_per_channel;

    auto dec_sample_num = blockIdx.x;
    auto dec_sample_offset = dec_sample_num * dm_rate;

    auto tap_offset = threadIdx.y * blockDim.y + threadIdx.x;

    hipComplex sample;
    if ((dec_sample_offset + threadIdx.x) >= samples_per_channel) {
        sample = make_hipComplex(0.0,0.0);
    }
    else {
        auto final_offset = channel_offset + dec_sample_offset + threadIdx.x;
        sample = original_samples[final_offset];
    }


    filter_products[tap_offset] = hipCmulf(sample,filter_taps[tap_offset]);

    __syncthreads();

    auto total_sum = parallel_reduce(filter_products, tap_offset);

    if (threadIdx.x == 0) {
        channel_offset = channel_num * samples_per_channel/dm_rate;
        auto total_channels = blockDim.y;
        auto freq_offset = threadIdx.y * total_channels;
        auto total_offset = freq_offset + channel_offset + dec_sample_num;
        decimated_samples[total_offset] = total_sum;
    }
}

__global__ void decimate2048(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel)
{

    extern __shared__ hipComplex filter_products[];

    auto channel_num = blockIdx.y;
    auto channel_offset = channel_num * samples_per_channel;

    auto dec_sample_num = blockIdx.x;
    auto dec_sample_offset = dec_sample_num * dm_rate;

    auto tap_offset = threadIdx.y * blockDim.y + 2 * threadIdx.x;

    hipComplex sample_1;
    hipComplex sample_2;
    if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_channel) {
        sample_1 = make_hipComplex(0.0,0.0);
        sample_2 = make_hipComplex(0.0,0.0);
    }
    else {
        auto final_offset = channel_offset + dec_sample_offset + 2*threadIdx.x;
        sample_1 = original_samples[final_offset];
        sample_2 = original_samples[final_offset+1];
    }

    filter_products[tap_offset] = hipCmulf(sample_1,filter_taps[tap_offset]);
    filter_products[tap_offset+1] = hipCmulf(sample_2, filter_taps[tap_offset+1]);

    __syncthreads();

    auto total_sum = parallel_reduce(filter_products, tap_offset);
    if (threadIdx.x == 0) {
        channel_offset = channel_num * samples_per_channel/dm_rate;
        auto total_channels = blockDim.y;
        auto freq_offset = threadIdx.y * total_channels;
        auto total_offset = freq_offset + channel_offset + dec_sample_num;
        decimated_samples[total_offset] = total_sum;
    }
}

static dim3 create_grid(uint32_t num_samples, uint32_t dm_rate, uint32_t num_channels)
{
    auto num_blocks_x = num_samples/dm_rate;
    auto num_blocks_y = num_channels;
    auto num_blocks_z = 1;
    std::cout << "    Grid size: " << num_blocks_x << " x " << num_blocks_y << " x "
        << num_blocks_z << std::endl;
    dim3 dimGrid(num_blocks_x,num_blocks_y,num_blocks_z);

    return dimGrid;
}

static dim3 create_block(uint32_t num_taps, uint32_t num_freqs)
{
    auto num_threads_x = num_taps;
    auto num_threads_y = num_freqs;
    auto num_threads_z = 1;
    std::cout << "    Block size: " << num_threads_x << " x " << num_threads_y << " x "
        << num_threads_z << std::endl;
    dim3 dimBlock(num_threads_x,num_threads_y,num_threads_z);

    return dimBlock;
}

void decimate1024_wrapper(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel, uint32_t num_taps, uint32_t num_freqs,
    uint32_t num_channels, hipStream_t stream) {

    auto shr_mem_taps = num_freqs * num_taps * sizeof(hipComplex);
    std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

    auto dimGrid = create_grid(samples_per_channel, dm_rate, num_channels);
    auto dimBlock = create_block(num_taps,num_freqs);
    decimate1024<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
                filter_taps, dm_rate, samples_per_channel);

}

void decimate2048_wrapper(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel, uint32_t num_taps, uint32_t num_freqs,
    uint32_t num_channels, hipStream_t stream) {

    auto shr_mem_taps = num_freqs * num_taps * sizeof(hipComplex);
    std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

    auto dimGrid = create_grid(samples_per_channel, dm_rate, num_channels);
    auto dimBlock = create_block(num_taps/2, num_freqs);
    decimate2048<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
        filter_taps, dm_rate, samples_per_channel);
}
