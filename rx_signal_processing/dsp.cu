#include "hip/hip_runtime.h"
#include "digital_processing.hpp"
#include "utils/protobuf/sigprocpacket.pb.h"
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <hip/hip_complex.h>
#include <chrono>
#include <thread>
#include "multithreading.h"

__global__ void decimate1024(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel)
{

    extern __shared__ hipComplex filter_products[];

    auto channel_num = blockIdx.y;
    auto channel_offset = channel_num * samples_per_channel;

    auto dec_sample_num = blockIdx.x;
    auto dec_sample_offset = dec_sample_num * dm_rate;

    auto tap_offset = threadIdx.y * blockDim.y + threadIdx.x;

    hipComplex sample;
    if ((dec_sample_offset + threadIdx.x) >= samples_per_channel) {
        sample = make_hipComplex(0.0,0.0);
    }
    else {
        auto final_offset = channel_offset + dec_sample_offset + threadIdx.x;
        sample = original_samples[final_offset];
    }


    filter_products[threadIdx.x] = hipCmulf(sample,filter_taps[tap_offset]);

    __syncthreads();


    auto num_taps = blockDim.x;
    for (unsigned int s=num_taps/2; s>32; s>>=1) {
        if (tap_offset < s)
            filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                    filter_products[tap_offset + s]);
        __syncthreads();
    }
    if (tap_offset < 32){
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 32]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 16]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 8]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 4]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 2]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 1]);
    }

    if (threadIdx.x == 0) {
        channel_offset = channel_num * samples_per_channel/dm_rate;
        auto total_channels = blockDim.y;
        auto freq_offset = threadIdx.y * total_channels;
        auto total_offset = freq_offset + channel_offset + dec_sample_num;
        decimated_samples[total_offset] = filter_products[tap_offset];
    }
}

__global__ void decimate2048(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel)
{

    extern __shared__ hipComplex filter_products[];

    auto channel_num = blockIdx.y;
    auto channel_offset = channel_num * samples_per_channel;

    auto dec_sample_num = blockIdx.x;
    auto dec_sample_offset = dec_sample_num * dm_rate;

    auto tap_offset = threadIdx.y * blockDim.y + 2 * threadIdx.x;

    hipComplex sample_1;
    hipComplex sample_2;
    if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_channel) {
        sample_1 = make_hipComplex(0.0,0.0);
        sample_2 = make_hipComplex(0.0,0.0);
    }
    else {
        auto final_offset = channel_offset + dec_sample_offset + 2*threadIdx.x;
        sample_1 = original_samples[final_offset];
        sample_2 = original_samples[final_offset+1];
    }


    filter_products[threadIdx.x] = hipCmulf(sample_1,filter_taps[tap_offset]);
    filter_products[threadIdx.x+1] = hipCmulf(sample_2, filter_taps[tap_offset+1]);

    __syncthreads();

    auto half_num_taps = blockDim.x;
    for (unsigned int s=half_num_taps; s>32; s>>=1) {
        if (tap_offset < s)
            filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                    filter_products[tap_offset + s]);
        __syncthreads();
    }
    if (tap_offset < 32){
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 32]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 16]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 8]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 4]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 2]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 1]);
    }

/*    auto half_num_taps = blockDim.x;
    for (unsigned int s=half_num_taps; s>32; s>>=1) {
        if (tap_offset < s)
            filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                    filter_products[tap_offset + s]);
            filter_products[tap_offset+1] = hipCaddf(filter_products[tap_offset+1],
                                                        filter_products[tap_offset+1 + s]);
        __syncthreads();
    }
    if (tap_offset < 32){
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 32]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 16]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 8]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 4]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 2]);
        filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],
                                                filter_products[tap_offset + 1]);

        filter_products[tap_offset + 1] = hipCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 32]);
        filter_products[tap_offset + 1] = hipCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 16]);
        filter_products[tap_offset + 1] = hipCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 8]);
        filter_products[tap_offset + 1] = hipCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 4]);
        filter_products[tap_offset + 1] = hipCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 2]);
        filter_products[tap_offset + 1] = hipCaddf(filter_products[tap_offset + 1],
                                                    filter_products[tap_offset + 1 + 1]);
    }*/
    if (threadIdx.x == 0) {
        channel_offset = channel_num * samples_per_channel/dm_rate;
        auto total_channels = blockDim.y;
        auto freq_offset = threadIdx.y * total_channels;
        auto total_offset = freq_offset + channel_offset + dec_sample_num;
        decimated_samples[total_offset] = filter_products[tap_offset];
    }
}

dim3 create_grid(uint32_t num_samples, uint32_t dm_rate, uint32_t num_channels)
{
    auto num_blocks_x = num_samples/dm_rate;
    auto num_blocks_y = num_channels;
    auto num_blocks_z = 1;
    std::cout << "    Grid size: " << num_blocks_x << " x " << num_blocks_y << " x "
        << num_blocks_z << std::endl;
    dim3 dimGrid(num_blocks_x,num_blocks_y,num_blocks_z);

    return dimGrid;
}

dim3 create_block(uint32_t num_taps, uint32_t num_freqs)
{
    auto num_threads_x = num_taps;
    auto num_threads_y = num_freqs;
    auto num_threads_z = 1;
    std::cout << "    Block size: " << num_threads_x << " x " << num_threads_y << " x "
        << num_threads_z << std::endl;
    dim3 dimBlock(num_threads_x,num_threads_y,num_threads_z);

    return dimBlock;
}

std::vector<hipDeviceProp_t> get_gpu_properties()
{
    std::vector<hipDeviceProp_t> gpu_properties;
    int num_devices = 0;

    gpuErrchk(hipGetDeviceCount(&num_devices));

    for(int i=0; i< num_devices; i++) {
            hipDeviceProp_t properties;
            gpuErrchk(hipGetDeviceProperties(&properties, i));
            gpu_properties.push_back(properties);
    }

    return gpu_properties;
}


DigitalProcessing::DigitalProcessing(zmq::socket_t *ack_s, zmq::socket_t *timing_s,
                                        uint32_t sq_num, const char* shr_mem_name)
{

    sequence_num = sq_num;
    ack_socket = ack_s;
    timing_socket = timing_s;

    gpuErrchk(hipStreamCreate(&stream));
    gpuErrchk(hipEventCreate(&initial_start));
    gpuErrchk(hipEventCreate(&kernel_start));
    gpuErrchk(hipEventCreate(&stop));
    gpuErrchk(hipEventRecord(initial_start, stream));

    shr_mem = new SharedMemoryHandler(shr_mem_name);
    shr_mem->open_shr_mem();

}

void DigitalProcessing::allocate_and_copy_rf_samples(uint32_t total_samples)
{

    rf_samples_size = total_samples * sizeof(hipComplex);
    gpuErrchk(hipMalloc(&rf_samples, rf_samples_size));
    gpuErrchk(hipMemcpyAsync(rf_samples,shr_mem->get_shrmem_addr(), rf_samples_size, hipMemcpyHostToDevice, stream));

}

void DigitalProcessing::allocate_and_copy_first_stage_filters(void *taps, uint32_t total_taps)
{
    first_stage_bp_filters_size = total_taps * sizeof(hipComplex);
    gpuErrchk(hipMalloc(&first_stage_bp_filters, first_stage_bp_filters_size));
    gpuErrchk(hipMemcpyAsync(first_stage_bp_filters, taps,
                first_stage_bp_filters_size, hipMemcpyHostToDevice, stream));
}

void DigitalProcessing::allocate_and_copy_second_stage_filters(void *taps, uint32_t total_taps)
{
    second_stage_filters_size = total_taps * sizeof(hipComplex);
    gpuErrchk(hipMalloc(&second_stage_filters, second_stage_filters_size));
    gpuErrchk(hipMemcpyAsync(second_stage_filters, taps,
               second_stage_filters_size, hipMemcpyHostToDevice, stream));
}

void DigitalProcessing::allocate_and_copy_third_stage_filters(void *taps, uint32_t total_taps)
{
    third_stage_filters_size = total_taps * sizeof(hipComplex);
    gpuErrchk(hipMalloc(&third_stage_filters, third_stage_filters_size));
    gpuErrchk(hipMemcpyAsync(third_stage_filters, taps,
                third_stage_filters_size, hipMemcpyHostToDevice, stream));
}

void DigitalProcessing::allocate_first_stage_output(uint32_t first_stage_samples)
{
    first_stage_output_size = first_stage_samples * sizeof(hipComplex);
    gpuErrchk(hipMalloc(&first_stage_output, first_stage_output_size));
}

void DigitalProcessing::allocate_second_stage_output(uint32_t second_stage_samples)
{
    second_stage_output_size = second_stage_samples * sizeof(hipComplex);
    gpuErrchk(hipMalloc(&second_stage_output, second_stage_output_size));
}

void DigitalProcessing::allocate_third_stage_output(uint32_t third_stage_samples)
{
    third_stage_output_size = third_stage_samples * sizeof(hipComplex);
    gpuErrchk(hipMalloc(&third_stage_output, third_stage_output_size));
}

void DigitalProcessing::allocate_and_copy_host_output(uint32_t host_samples)
{
    host_output_size = host_samples * sizeof(hipComplex);
    gpuErrchk(hipHostAlloc(&host_output, host_output_size, hipHostMallocDefault));
    gpuErrchk(hipMemcpyAsync(host_output, third_stage_output,
                host_output_size, hipMemcpyDeviceToHost,stream));
}

void DigitalProcessing::copy_output_to_host()
{
    gpuErrchk(hipMemcpy(host_output, third_stage_output,
               host_output_size, hipMemcpyDeviceToHost));
}

void DigitalProcessing::clear_device_and_destroy()
{
    gpuErrchk(hipFree(rf_samples));
    gpuErrchk(hipFree(first_stage_bp_filters));
    gpuErrchk(hipFree(second_stage_filters));
    gpuErrchk(hipFree(third_stage_filters));
    gpuErrchk(hipFree(first_stage_output));
    gpuErrchk(hipFree(second_stage_output));
    gpuErrchk(hipFree(third_stage_output));
    gpuErrchk(hipHostFree(host_output));
    gpuErrchk(hipEventDestroy(initial_start));
    gpuErrchk(hipEventDestroy(kernel_start));
    gpuErrchk(hipEventDestroy(stop));
    gpuErrchk(hipStreamDestroy(stream));

    shr_mem->remove_shr_mem();
    delete shr_mem;

}

void DigitalProcessing::stop_timing()
{
    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));

    gpuErrchk(hipEventElapsedTime(&total_process_timing_ms, initial_start, stop));
    gpuErrchk(hipEventElapsedTime(&decimate_kernel_timing_ms, kernel_start, stop));

}

void DigitalProcessing::send_timing()
{
    sigprocpacket::SigProcPacket sp;
    sp.set_kerneltime(decimate_kernel_timing_ms);
    sp.set_sequence_num(sequence_num);

    std::string s_msg_str;
    sp.SerializeToString(&s_msg_str);
    zmq::message_t s_msg(s_msg_str.size());
    memcpy ((void *) s_msg.data (), s_msg_str.c_str(), s_msg_str.size());

    timing_socket->send(s_msg);
    std::cout << "Sent timing after processing" << std::endl;

}

void postprocess(DigitalProcessing *dp)
{

    dp->stop_timing();
    dp->send_timing();
    std::cout << "Cuda kernel timing: " << dp->get_decimate_timing()
        << "ms" <<std::endl;
    std::cout << "Complete process timing: " << dp->get_total_timing()
        << "ms" <<std::endl;

    dp->clear_device_and_destroy();
    delete dp;
}


void CUDART_CB DigitalProcessing::cuda_postprocessing_callback(hipStream_t stream, hipError_t status,
                                                        void *processing_data)
{
    gpuErrchk(status);
    std::thread start_pp(postprocess,static_cast<DigitalProcessing*>(processing_data));
    start_pp.detach();
}


void DigitalProcessing::send_ack()
{
    sigprocpacket::SigProcPacket sp;
    sp.set_sequence_num(sequence_num);

    std::string s_msg_str;
    sp.SerializeToString(&s_msg_str);
    zmq::message_t s_msg(s_msg_str.size());
    memcpy ((void *) s_msg.data (), s_msg_str.c_str(), s_msg_str.size());
    ack_socket->send(s_msg);
    std::cout << "Sent ack after copy" << std::endl;
}

void DigitalProcessing::start_decimate_timing()
{
    gpuErrchk(hipEventRecord(kernel_start, stream));
}

void initial_memcpy_callback_handler(DigitalProcessing *dp)
{
    dp->send_ack();
    dp->start_decimate_timing();
}

void CUDART_CB DigitalProcessing::initial_memcpy_callback(hipStream_t stream, hipError_t status,
                                                void *processing_data)
{
    gpuErrchk(status);
    std::thread start_imc(initial_memcpy_callback_handler,
                            static_cast<DigitalProcessing*>(processing_data));
    start_imc.join();

}

void DigitalProcessing::call_decimate(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel, uint32_t num_taps, uint32_t num_freqs,
    uint32_t num_channels, const char *output_msg) {

    std::cout << output_msg << std::endl;


    auto gpu_properties = get_gpu_properties();
    auto shr_mem_taps = num_taps * sizeof(hipComplex);
    std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

    auto dimGrid = create_grid(samples_per_channel, dm_rate, num_channels);



    //For now we have a kernel that will process 2 samples per thread if need be
    if (num_taps * num_freqs > 2 * gpu_properties[0].maxThreadsPerBlock) {
        //TODO(Keith) : handle error
    }
    else if (num_taps * num_freqs > gpu_properties[0].maxThreadsPerBlock) {
        auto dimBlock = create_block(num_taps/2, num_freqs);
        decimate2048<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
            filter_taps, dm_rate, samples_per_channel);
    }
    else {
        auto dimBlock = create_block(num_taps,num_freqs);
        decimate1024<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
            filter_taps, dm_rate, samples_per_channel);;
    }
    gpuErrchk(hipPeekAtLastError());

}

hipComplex* DigitalProcessing::get_rf_samples_p(){
    return rf_samples;
}

hipComplex* DigitalProcessing::get_first_stage_bp_filters_p(){
    return first_stage_bp_filters;
}

hipComplex* DigitalProcessing::get_second_stage_filters_p(){
    return second_stage_filters;
}

hipComplex* DigitalProcessing::get_third_stage_filters_p(){
    return third_stage_filters;
}

hipComplex* DigitalProcessing::get_first_stage_output_p(){
    return first_stage_output;
}

hipComplex* DigitalProcessing::get_second_stage_output_p(){
    return second_stage_filters;
}

hipComplex* DigitalProcessing::get_third_stage_output_p(){
    return third_stage_filters;
}

hipStream_t DigitalProcessing::get_cuda_stream(){
    return stream;
}

float DigitalProcessing::get_total_timing()
{
    return total_process_timing_ms;
}

float DigitalProcessing::get_decimate_timing()
{
    return decimate_kernel_timing_ms;
}



/*uint32_t DigitalProcessing::get_sequence_num()
{
    return sequence_num;
}

zmq::socket_t* DigitalProcessing::get_rctl_socket()
{
    return rctl_socket;
}

zmq::socket_t* DigitalProcessing::get_timing_socket()
{
    return timing_socket;
}*/



